#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * License); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * AS IS BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*
 * Copyright (c) 2021, OPEN AI LAB
 * Author: hhchen@openailab.com
 */


#include "cuda_executor.hpp"

extern "C"
{
#include "tengine_op.h"
#include "eltwise_param.h"
}

__global__ void eltwise_sum(float *y, float *x0, float *x1, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
    {
        y[idx] = x0[idx] + x1[idx];
    }
}

void eltwisesum_gpu_kernel(struct ir_graph* ir_graph, struct ir_node* ir_node, dict_uint2voidx  gpu_addr_map)
{
    struct ir_tensor* input_tensor0 = get_ir_graph_tensor(ir_graph, ir_node->input_tensors[0]);
    struct ir_tensor* input_tensor1 = get_ir_graph_tensor(ir_graph, ir_node->input_tensors[1]);
    struct ir_tensor* output_tensor = get_ir_graph_tensor(ir_graph, ir_node->output_tensors[0]);

    eltwise_param* param = (eltwise_param*)ir_node->op.param_mem;

    /* init grid and block */
    int bs = 1024;
    int s = ceil((output_tensor->elem_num + bs - 1.) / bs);
    dim3 grid = dim3(s);

    switch (param->type)
    {
        case ELT_SUM:
            eltwise_sum<<<grid, bs>>>((float*)gpu_addr_map[output_tensor->idx], (float*)gpu_addr_map[input_tensor0->idx], (float*)gpu_addr_map[input_tensor1->idx], output_tensor->elem_num);
            break;
        default:
            break;
    }
}

void CUDAEngine::AddEltwiseNode(struct ir_graph* ir_graph, struct ir_node* ir_node)
{
    TLOG_INFO("Tengine GPU: Support OP(%d) OP_RELU.\n", ir_node->idx);
    eltwisesum_gpu_kernel(ir_graph, ir_node, this->gpu_addr_map);
    this->ops.push_back(std::bind(&eltwisesum_gpu_kernel, ir_graph, ir_node, this->gpu_addr_map));
}
