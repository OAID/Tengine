#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * License); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * AS IS BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*
 * Copyright (c) 2021, OPEN AI LAB
 * Author: hhchen@openailab.com
 */


#include "cuda_executor.hpp"

extern "C"
{
#include "slice_param.h"

#include "graph/tensor.h"
#include "operator/op.h"
#include "utility/log.h"
}

__global__ void slice(float *y, float *x, int elem_num, int res)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    int idx_new = idx + res;

    if (idx < elem_num)
    {
        y[idx] = x[idx_new];
    }
}

void slice_gpu_kernel(struct graph* ir_graph, struct node* ir_node, dict_uint2voidx  gpu_addr_map)
{
    struct tensor* input_tensor = get_ir_graph_tensor(ir_graph, ir_node->input_tensors[0]);
    struct tensor* output_tensor = get_ir_graph_tensor(ir_graph, ir_node->output_tensors[0]);

    /* init grid and block */
    int bs = 1024;
    int s = ceil((output_tensor->elem_num + bs - 1.) / bs);
    dim3 grid = dim3(s);

    struct slice_param* param = (struct slice_param*)ir_node->op.param_mem;
    int res = 1;
    for (uint8_t i = input_tensor->dim_num-1; i > param->axis; i--)
    {
        res *= input_tensor->dims[i];
    }
    res *= param->begin;

    slice<<<grid, bs>>>((float*)gpu_addr_map[output_tensor->index], (float*)gpu_addr_map[input_tensor->index], output_tensor->elem_num, res);
}

void CUDAEngine::AddSliceNode(struct graph* ir_graph, struct node* ir_node)
{
    TLOG_INFO("Tengine GPU: Support OP(%d) OP_SLICE.\n", ir_node->index);
    slice_gpu_kernel(ir_graph, ir_node, this->gpu_addr_map);
    this->ops.push_back(std::bind(&slice_gpu_kernel, ir_graph, ir_node, this->gpu_addr_map));
}
