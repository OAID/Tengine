#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * License); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * AS IS BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*
 * Copyright (c) 2021, OPEN AI LAB
 * Author: hhchen@openailab.com
 */
 

#include "cuda_executor.hpp"

extern "C"
{
#include "clip_param.h"

#include "graph/tensor.h"
#include "operator/op.h"
#include "utility/log.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__ void relu6(float *y, float *x, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
    {
        y[idx] = x[idx] > 6 ? 6 : x[idx];
        y[idx] = y[idx] < 0 ? 0 : y[idx];
    }
}

void relu6_gpu_kernel(struct graph* ir_graph, struct node* ir_node, dict_uint2voidx  gpu_addr_map)
{
    struct tensor* input_tensor = get_ir_graph_tensor(ir_graph, ir_node->input_tensors[0]);
    struct tensor* output_tensor = get_ir_graph_tensor(ir_graph, ir_node->output_tensors[0]);

    /* init grid and block */
    int bs = 1024;
    int s = ceil((output_tensor->elem_num + bs - 1.) / bs);
    dim3 grid = dim3(s);

    relu6<<<grid, bs>>>((float*)gpu_addr_map[output_tensor->index], (float*)gpu_addr_map[input_tensor->index], output_tensor->elem_num);
}

void CUDAEngine::AddClipNode(struct graph* ir_graph, struct node* ir_node)
{
    TLOG_INFO("Tengine GPU: Support OP(%d) OP_CLIP.\n", ir_node->index);
    relu6_gpu_kernel(ir_graph, ir_node, this->gpu_addr_map);
    this->ops.push_back(std::bind(&relu6_gpu_kernel, ir_graph, ir_node, this->gpu_addr_map));
}
