#include "hip/hip_runtime.h"


#include "cuda_executor.hpp"

extern "C"
{
#include "convolution_param.h"

#include "graph/tensor.h"
#include "operator/op.h"
#include "utility/log.h"
}

__global__ void bias_add(float *y, float *x, int elem_num_perimg, int elem_perchannel, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
    {
        y[idx] += x[idx % elem_num_perimg / elem_perchannel];
    }
}

__global__ void bias_add_relu(float *y, float *x, int elem_num_perimg, int elem_perchannel, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
    {
        y[idx] += x[idx % elem_num_perimg / elem_perchannel];
        y[idx] = y[idx] > 0 ? y[idx] : 0;
    }
}

void conv_gpu_kernel(hipdnnHandle_t& handle, struct graph* ir_graph, struct node* ir_node, dict_uint2voidx  gpu_addr_map,
                     hipdnnConvolutionFwdAlgo_t& algo1, int setalgo)
{
    struct tensor* conv_input_data = get_ir_graph_tensor(ir_graph, ir_node->input_tensors[0]);
    struct tensor* conv_weight = get_ir_graph_tensor(ir_graph, ir_node->input_tensors[1]);
    struct tensor* conv_output_data = get_ir_graph_tensor(ir_graph, ir_node->output_tensors[0]);

    struct conv_param* conv_param = ( struct conv_param* )ir_node->op.param_mem;

    // input
    hipdnnTensorDescriptor_t input_descriptor;
    hipdnnCreateTensorDescriptor(&input_descriptor);
    hipdnnSetTensor4dDescriptor(input_descriptor,
                               HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT,
                               conv_input_data->dims[0], conv_input_data->dims[1], conv_input_data->dims[2], conv_input_data->dims[3]);

    // output
    hipdnnTensorDescriptor_t output_descriptor;
    hipdnnCreateTensorDescriptor(&output_descriptor);
    hipdnnSetTensor4dDescriptor(output_descriptor,
                               HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT,
                               conv_output_data->dims[0], conv_output_data->dims[1], conv_output_data->dims[2], conv_output_data->dims[3]);

    // kernel
    hipdnnFilterDescriptor_t kernel_descriptor;
    hipdnnCreateFilterDescriptor(&kernel_descriptor);
    hipdnnSetFilter4dDescriptor(kernel_descriptor,
                               HIPDNN_DATA_FLOAT,
                               HIPDNN_TENSOR_NCHW,
                               conv_weight->dims[0], conv_weight->dims[1], conv_weight->dims[2], conv_weight->dims[3]);

    // convolution descriptor
    hipdnnConvolutionDescriptor_t conv_descriptor;
    hipdnnCreateConvolutionDescriptor(&conv_descriptor);
    hipdnnSetConvolution2dDescriptor(conv_descriptor,
                                    conv_param->pad_h0, conv_param->pad_w0, // zero-padding
                                    conv_param->stride_h, conv_param->stride_w, // stride
                                    conv_param->dilation_h, conv_param->dilation_w,
                                    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);
    hipdnnSetConvolutionGroupCount(conv_descriptor, conv_param->group);

//    // algorithm
//    hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_GEMM;
//
//    // workspace size && allocate memory
//    size_t workspace_size;
//    hipdnnGetConvolutionForwardWorkspaceSize(
//            handle, input_descriptor, kernel_descriptor, conv_descriptor, output_descriptor, algo, &workspace_size);
//
//    size_t workspace_size;
//    hipdnnGetConvolutionForwardWorkspaceSize(handle,
//                                            input_descriptor,
//                                            kernel_descriptor,
//                                            conv_descriptor,
//                                            output_descriptor,
//                                            algo,
//                                            &workspace_size);
//
//    void * workspace = nullptr;
//    hipMalloc(&workspace, workspace_size);

    if (0 == setalgo)
    {
        int returnedAlgoCount;
        hipdnnConvolutionFwdAlgoPerf_t algo;
        auto ret0 = hipdnnFindConvolutionForwardAlgorithm(handle,
                                                         input_descriptor,
                                                         kernel_descriptor,
                                                         conv_descriptor,
                                                         output_descriptor,
                                                         1,
                                                         &returnedAlgoCount,
                                                         &algo
        );
        algo1 = algo.algo;
    }
    size_t workspace_size;
    auto ret1 = hipdnnGetConvolutionForwardWorkspaceSize(
        handle, input_descriptor, kernel_descriptor, conv_descriptor, output_descriptor, algo1, &workspace_size);
    void * workspace = nullptr;
    hipMalloc(&workspace, workspace_size);


//    size_t workspace_size = 0;
//    void * workspace = nullptr;
//    hipMalloc(&workspace, workspace_size);
//    if (0 == setalgo)
//    {
//        int returnedAlgoCount;
//        hipdnnConvolutionFwdAlgoPerf_t algo;
//        auto ret0 = hipdnnFindConvolutionForwardAlgorithmEx(handle,
//                                                           input_descriptor, gpu_addr_map[conv_input_data->index],
//                                                           kernel_descriptor, gpu_addr_map[conv_weight->index],
//                                                           conv_descriptor,
//                                                           output_descriptor, gpu_addr_map[conv_output_data->index],
//                                                           1,
//                                                           &returnedAlgoCount,
//                                                           &algo,
//                                                           workspace,
//                                                           workspace_size
//                                                           );
//        algo1 = algo.algo;
//    }


    /* convolution forward run */
    auto alpha = 1.0f, beta = 0.0f;
    auto ret2 = hipdnnConvolutionForward(handle,
                                        &alpha, input_descriptor, gpu_addr_map[conv_input_data->index],
                                        kernel_descriptor, gpu_addr_map[conv_weight->index],
                                        conv_descriptor, algo1,
                                        workspace, workspace_size,
                                        &beta, output_descriptor, gpu_addr_map[conv_output_data->index]);

    /* init grid and block */
    int bs = 1024;
    int s = ceil((conv_output_data->elem_num + bs - 1.) / bs);
    dim3 grid = dim3(s);

    /* bias_add and relu compute */
    if (2 < ir_node->input_num)
    {
        struct tensor* conv_bias = get_ir_graph_tensor(ir_graph, ir_node->input_tensors[2]);
        int elem_23 = conv_output_data->dims[2] * conv_output_data->dims[3];
        int elem_123 = conv_output_data->dims[1] * elem_23;

        if (conv_param->activation == 0)
            bias_add_relu<<<grid, bs>>>((float*)gpu_addr_map[conv_output_data->index], (float*)gpu_addr_map[conv_bias->index], elem_123, elem_23, conv_output_data->elem_num);
        else
            bias_add<<<grid, bs>>>((float*)gpu_addr_map[conv_output_data->index], (float*)gpu_addr_map[conv_bias->index], elem_123, elem_23, conv_output_data->elem_num);
    }

    hipFree(&workspace);
}



void CUDAEngine::AddConvolutionNode(struct graph* ir_graph, struct node* ir_node)
{
    TLOG_INFO("Tengine GPU: Support OP(%d) OP_CONV.\n", ir_node->index);
    hipdnnCreate(&this->cudnn_handle);
    conv_gpu_kernel(this->cudnn_handle, ir_graph, ir_node, this->gpu_addr_map, this->algo1, 0);
    this->ops.push_back(std::bind(&conv_gpu_kernel, this->cudnn_handle, ir_graph, ir_node, this->gpu_addr_map, this->algo1, 1));
}
