#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * License); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * AS IS BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*
 * Copyright (c) 2021, OPEN AI LAB
 * Author: hhchen@openailab.com
 */


#include "cuda_executor.hpp"

extern "C"
{
#include "eltwise_param.h"

#include "graph/tensor.h"
#include "operator/op.h"
#include "utility/log.h"
}

__global__ void eltwise_sum(float *y, float *x0, float *x1, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
    {
        y[idx] = x0[idx] + x1[idx];
    }
}

void eltwisesum_gpu_kernel(struct graph* ir_graph, struct node* ir_node, dict_uint2voidx  gpu_addr_map)
{
    struct tensor* input_tensor0 = get_ir_graph_tensor(ir_graph, ir_node->input_tensors[0]);
    struct tensor* input_tensor1 = get_ir_graph_tensor(ir_graph, ir_node->input_tensors[1]);
    struct tensor* output_tensor = get_ir_graph_tensor(ir_graph, ir_node->output_tensors[0]);

    eltwise_param* param = (eltwise_param*)ir_node->op.param_mem;

    /* init grid and block */
    int bs = 1024;
    int s = ceil((output_tensor->elem_num + bs - 1.) / bs);
    dim3 grid = dim3(s);

    switch (param->type)
    {
        case ELT_SUM:
            eltwise_sum<<<grid, bs>>>((float*)gpu_addr_map[output_tensor->index], (float*)gpu_addr_map[input_tensor0->index], (float*)gpu_addr_map[input_tensor1->index], output_tensor->elem_num);
            break;
        default:
            break;
    }
}

void CUDAEngine::AddEltwiseNode(struct graph* ir_graph, struct node* ir_node)
{
    TLOG_INFO("Tengine GPU: Support OP(%d) OP_ELTWISE.\n", ir_node->index);
    eltwisesum_gpu_kernel(ir_graph, ir_node, this->gpu_addr_map);
    this->ops.push_back(std::bind(&eltwisesum_gpu_kernel, ir_graph, ir_node, this->gpu_addr_map));
}
