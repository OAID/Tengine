#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * License); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * AS IS BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*
 * Copyright (c) 2021, OPEN AI LAB
 * Author: hhchen@openailab.com
 */


#include "cuda_executor.hpp"

extern "C"
{
#include "permute_param.h"

#include "graph/tensor.h"
#include "operator/op.h"
#include "utility/log.h"
}

__global__ void permute(float *y, float *x, int elem_num, int n, int c, int hw)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    int chw = c * hw;
    int idx_n = idx / chw;
    int idx_c = idx % chw / hw;
    int idx_hw = idx % hw;

    int idx_new = idx_n * chw + idx_hw * c + idx_c ;

    if (idx < elem_num)
    {
        y[idx_new] = x[idx];
    }
}

void permute_gpu_kernel(struct graph* ir_graph, struct node* ir_node, dict_uint2voidx  gpu_addr_map)
{
    struct tensor* input_tensor = get_ir_graph_tensor(ir_graph, ir_node->input_tensors[0]);
    struct tensor* output_tensor = get_ir_graph_tensor(ir_graph, ir_node->output_tensors[0]);

    struct permute_param* param = (struct permute_param*)ir_node->op.param_mem;

    /* init grid and block */
    int bs = 1024;
    int s = ceil((output_tensor->elem_num + bs - 1.) / bs);
    dim3 grid = dim3(s);

    if (param->order0 == 0 && param->order1 == 2 && param->order2 == 3 && param->order3 == 1)
        permute<<<grid, bs>>>((float*)gpu_addr_map[output_tensor->index], (float*)gpu_addr_map[input_tensor->index], output_tensor->elem_num,
                              input_tensor->dims[0], input_tensor->dims[1], input_tensor->dims[2] * input_tensor->dims[3]);
}

void CUDAEngine::AddPermuteNode(struct graph* ir_graph, struct node* ir_node)
{
    TLOG_INFO("Tengine GPU: Support OP(%d) OP_PERMUTE.\n", ir_node->index);
    permute_gpu_kernel(ir_graph, ir_node, this->gpu_addr_map);
    this->ops.push_back(std::bind(&permute_gpu_kernel, ir_graph, ir_node, this->gpu_addr_map));
}
